#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "/home/user14/Vishwjeet_Ujgare_Feb_2024/vishwjeet_project/Image-Processing/external_lib/stb/stb_image_write.h"
#include "/home/user14/Vishwjeet_Ujgare_Feb_2024/vishwjeet_project/Image-Processing/external_lib/stb/stb_image.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(unsigned char* d_in, unsigned char* d_out, int width, int height, int channels) {
    int x = threadIdx.x + blockIdx.x * blockDim.x; // cols
    int y = threadIdx.y + blockIdx.y * blockDim.y; // rows

    if (y < height && x < width) {
        int greyOffset = y * width + x;
        int bgrOffset = greyOffset * channels;
        unsigned char b = d_in[bgrOffset];
        unsigned char g = d_in[bgrOffset + 1];
        unsigned char r = d_in[bgrOffset + 2];

        d_out[greyOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}

int main() {
    // const char* path = "D:\\Cuda_prac\\Image\\1280x720.jpg"; // hardcoding the path

  const char* path = "/home/user14/Vishwjeet_Ujgare_Feb_2024/vishwjeet_project/Image-Processing/data/input/4k-3840-x-2160-wallpapers-themefoxx.jpg";
    // Load the image using stb_image
    int width, height, channels;
    unsigned char *h_image = stbi_load(path, &width, &height, &channels, 0);
    if (!h_image) {
        std::cerr << "Failed to find or read " << path << std::endl;
        return -1;
    }

    // Allocate memory on the GPU
    unsigned char *d_in, *d_out;
    hipMalloc((void**)&d_in, width * height * channels * sizeof(unsigned char));
    hipMalloc((void**)&d_out, width * height * sizeof(unsigned char));

    // Copy image data to the GPU
    hipMemcpy(d_in, h_image, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Setting block and grid dimensions
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    // Launching the kernel
    kernel<<<dimGrid, dimBlock>>>(d_in, d_out, width, height, channels);
    hipDeviceSynchronize();

    // Allocate host memory for the output image
    unsigned char *h_image_processed = (unsigned char*)malloc(width * height * sizeof(unsigned char));

    // Copy the processed image data back to the host
    hipMemcpy(h_image_processed, d_out, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save the processed image using stb_image_write
    stbi_write_jpg("/home/user14/Vishwjeet_Ujgare_Feb_2024/vishwjeet_project/Image-Processing/data/output/cuda_processed_img/Processed_Image.jpg", width, height, 1, h_image_processed, 100);

    // Free the allocated memory
    stbi_image_free(h_image);
    free(h_image_processed);
    hipFree(d_in);
    hipFree(d_out);

    std::cout << "Image processed and saved as Processed_Image.jpg" << std::endl;

    return 0;
}
